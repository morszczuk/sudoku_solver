#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

void addArray(int* a, int size, int* res)
{
	int i = 0;
	for (i = 1; i < size; i++)
	{
		a[0] += a[i];
	}
	
	*res = a[0];
}

__global__ void kernel(int* a, int size, int* res)
{
	int idx= blockDim.x*blockIdx.x + threadIdx.x;

	for (int i = 1; i <= size / 2; i *= 2)
	{
		if (idx % (2 * i) == 0) {
			printf("BEFORE [Thread %d]: %d\n", idx, a[idx]);
			a[idx] += a[idx + i];
			printf("AFTER [Thread %d]: %d\n", idx, a[idx]);
		}
		else
		{
			printf("[Thread %d] returning\n", idx);
			return;
		}
		__syncthreads();
	}         

	*res = a[idx];
}

int main()
{
	int N = 8;
	int *h_arr, *d_arr, *h_res, *d_res;
	hipEvent_t start, stop;
	float time;


	h_arr = (int*)malloc(N * sizeof(int));
	h_res = (int*)malloc(sizeof(int));
	hipMalloc(&d_arr, N*sizeof(int));
	hipMalloc(&d_res, sizeof(int));

	for (int i = 0; i < N; i++)
	{
		h_arr[i] = i;
		printf("[%d]", i);
	}

	hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	kernel<<<1, 8>>>(d_arr, N, d_res);

	hipMemcpy(h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);

	//addArray(h_arr, 8, h_res);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	printf("Result: %d, Time: %f", *h_res, time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	getchar();

	return 0;
}
