
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include <stdio.h>
//#include <stdlib.h>
//#include <fstream>
//#include <string>
//#include <sstream>
//
//__global__ void checkCorrectness(int data[9][9], int* d_number_presence)
//{
//	extern __shared__ int number_presence[];
//	int idx = blockDim.y*blockIdx.y + threadIdx.y;
//	int idy = blockDim.x*blockIdx.x + threadIdx.x;
//	int index_1, index_2, index_3;
//	int k = 81;
//
//	//printf("[Thread pos: %d | %d, val: %d]\n", idx, idy, data[idx][idy]);
//
//	number_presence[idx * 9 + idy] = 0;
//	number_presence[k + idx * 9 + idy] = 0;
//	number_presence[(2*k) + (idx * 9 + idy)] = 0;
//
//	index_1 = idx * 9 + data[idx][idy] - 1;
//	index_2 = k + idy * 9 + data[idx][idy] - 1;
//	index_3 = (2 * k) + ((idx / 3) * 27) + ((idy / 3) * 9) + data[idx][idy] - 1;
//
//	printf("[idx: %d, idy: %d | val: %d | %d, %d, %d]\n", idx, idy, data[idx][idy], index_1, index_2 - 81 , index_3 - 162);
//	//printf("%d, %d, %d\n", index_1, index_2 - 81, index_3 - 162);
//	
//	__syncthreads();
//
//	if (data[idx][idy] > 0)
//	{
//		number_presence[idx * 9 + data[idx][idy] - 1] = 1; //informs, is number in data[idx][idy] - 1 is present in row idx
//		number_presence[k + (idy * 9 + data[idx][idy] - 1)] = 1; //informs, is number in data[idx][idy] - 1 is present in column idy
//		number_presence[(2 * k) + ((idx / 3) * 27) + ((idy / 3) * 9) + data[idx][idy] - 1] = 1; //informs, that number which is in data[idx][idy] - 1 is present in proper 'quarter'
//	}
//
//	__syncthreads();
//
//	d_number_presence[idx * 9 + idy] = number_presence[idx * 9 + idy];
//	d_number_presence[k + idx * 9 + idy] = number_presence[k + idx * 9 + idy];
//	d_number_presence[(2 * k) + (idx * 9 + idy)] = number_presence[(2 * k) + (idx * 9 + idy)];
//
//
//	//number_presence[idx][data[idx][idy] - 1] = (data[idx][idy] > 0) ? 1 : 0;
//
//	//__syncthreads();
//
//	//number_presence[9 + idy][data[idx][idy] - 1] = (data[idx][idy] > 0) ? 1 : 0;
//
//	//__syncthreads();
//
//	//d_number_presence[idx][data[idx][idy] - 1] = number_presence[idx][data[idx][idy] - 1];
//	//d_number_presence[9 + idy][data[idx][idy] - 1] = number_presence[9 + idy][data[idx][idy] - 1];
//
//	//__syncthreads();
//}
//
//__global__ void addArray(int* a, int size, int* res)
//{
//	int idx = blockDim.x*blockIdx.x + threadIdx.x;
//
//	for (int i = 1; i <= size / 2; i *= 2)
//	{
//		if (idx % (2 * i) == 0) {
//			printf("BEFORE [Thread %d]: %d\n", idx, a[idx]);
//			a[idx] += a[idx + i];
//			printf("AFTER [Thread %d]: %d\n", idx, a[idx]);
//		}
//		else
//		{
//			printf("[Thread %d] returning\n", idx);
//			return;
//		}
//		__syncthreads();
//	}
//
//	*res = a[idx];
//}
//
//void setValues(int** h_sudoku)
//{
//	h_sudoku[0][0] = 2;
//	h_sudoku[0][1] = 0;
//	h_sudoku[0][2] = 0;
//	h_sudoku[0][3] = 8;
//	h_sudoku[0][4] = 0;
//	h_sudoku[0][5] = 4;
//	h_sudoku[0][6] = 0;
//	h_sudoku[0][7] = 0;
//	h_sudoku[0][8] = 6;
//
//	h_sudoku[1][0] = 0;
//	h_sudoku[1][1] = 0;
//	h_sudoku[1][2] = 6;
//	h_sudoku[1][3] = 0;
//	h_sudoku[1][4] = 0;
//	h_sudoku[1][5] = 0;
//	h_sudoku[1][6] = 5;
//	h_sudoku[1][7] = 0;
//	h_sudoku[1][8] = 0;
//
//	h_sudoku[2][0] = 0;
//	h_sudoku[2][1] = 7;
//	h_sudoku[2][2] = 4;
//	h_sudoku[2][3] = 0;
//	h_sudoku[2][4] = 0;
//	h_sudoku[2][5] = 0;
//	h_sudoku[2][6] = 9;
//	h_sudoku[2][7] = 2;
//	h_sudoku[2][8] = 0;
//
//	h_sudoku[3][0] = 3;
//	h_sudoku[3][1] = 0;
//	h_sudoku[3][2] = 0;
//	h_sudoku[3][3] = 0;
//	h_sudoku[3][4] = 4;
//	h_sudoku[3][5] = 0;
//	h_sudoku[3][6] = 0;
//	h_sudoku[3][7] = 0;
//	h_sudoku[3][8] = 7;
//
//	h_sudoku[4][0] = 0;
//	h_sudoku[4][1] = 0;
//	h_sudoku[4][2] = 0;
//	h_sudoku[4][3] = 3;
//	h_sudoku[4][4] = 0;
//	h_sudoku[4][5] = 5;
//	h_sudoku[4][6] = 0;
//	h_sudoku[4][7] = 0;
//	h_sudoku[4][8] = 0;
//
//	h_sudoku[5][0] = 4;
//	h_sudoku[5][1] = 0;
//	h_sudoku[5][2] = 0;
//	h_sudoku[5][3] = 0;
//	h_sudoku[5][4] = 6;
//	h_sudoku[5][5] = 0;
//	h_sudoku[5][6] = 0;
//	h_sudoku[5][7] = 0;
//	h_sudoku[5][8] = 9;
//
//	h_sudoku[6][0] = 0;
//	h_sudoku[6][1] = 1;
//	h_sudoku[6][2] = 9;
//	h_sudoku[6][3] = 0;
//	h_sudoku[6][4] = 0;
//	h_sudoku[6][5] = 0;
//	h_sudoku[6][6] = 7;
//	h_sudoku[6][7] = 4;
//	h_sudoku[6][8] = 0;
//
//	h_sudoku[7][0] = 0;
//	h_sudoku[7][1] = 0;
//	h_sudoku[7][2] = 8;
//	h_sudoku[7][3] = 0;
//	h_sudoku[7][4] = 0;
//	h_sudoku[7][5] = 0;
//	h_sudoku[7][6] = 2;
//	h_sudoku[7][7] = 0;
//	h_sudoku[7][8] = 0;
//
//	h_sudoku[8][0] = 5;
//	h_sudoku[8][1] = 0;
//	h_sudoku[8][2] = 0;
//	h_sudoku[8][3] = 6;
//	h_sudoku[8][4] = 0;
//	h_sudoku[8][5] = 8;
//	h_sudoku[8][6] = 0;
//	h_sudoku[8][7] = 0;
//	h_sudoku[8][8] = 1;
//}
//
//int** readSudokuArray(char* filename)
//{
//	int** h_sudoku = new int*[9];
//
//	for (int i = 0; i < 9; i++) {
//		h_sudoku[i] = new int[9];
//	}
//
//	//printf("SUDOKU FILENAME: %s\n", filename);
//	std::ifstream sudoku_file(filename);
//
//	int a0, a1, a2, a3, a4, a5, a6, a7, a8;
//	int i = 0;
//
//	while (sudoku_file >> a0 >> a1 >> a2 >> a3 >> a4 >> a5 >> a6 >> a7 >> a8)
//	{
//		h_sudoku[i][0] = a0;
//		h_sudoku[i][1] = a1;
//		h_sudoku[i][2] = a2;
//		h_sudoku[i][3] = a3;
//		h_sudoku[i][4] = a4;
//		h_sudoku[i][5] = a5;
//		h_sudoku[i][6] = a6;
//		h_sudoku[i][7] = a7;
//		h_sudoku[i][8] = a8;
//		i++;
//	}
//
//	return h_sudoku;
//}
//
//void printArray(int** h_sudoku, int N, int M)
//{
//	for (int i = 0; i < N; i++)
//	{
//		for (int j = 0; j < M; j++)
//			printf("%d |", h_sudoku[i][j]);
//		printf("\n");
//
//		for (int j = 0; j < N; j++)
//			printf("- |");
//		printf("\n");
//	}
//}
//
//void printOneDimArray(int* arr, int N, int M)
//{
//	for (int i = 0; i < N; i++)
//	{
//		for (int j = 0; j < M; j++)
//		{
//			printf("%d|", arr[i*N + j]);
//		}
//		printf("\n");
//		if (i != 0 && i % 9 == 0)
//			printf("\n");
//	}
//}
//
//int main()
//{
//	int N = 9;
//	int ** h_sudoku;
//	int * h_number_presence = new int[243];
//	int * h_number_presence_nice_adding = new int[256];
//
//	int *h_result;
//	int(*d_sudoku)[9];
//	int* d_number_presence;
//	int* d_number_presence_nice_adding;
//
//	int *d_result;
//	
//	char filename[] = "arr_1_solved.txt";
//	//char filename[] = "arr_1_unsolved.txt";
//	int help[9][9];
//	int sharedMemorySize;
//	
//	cudaEvent_t start, stop;
//	float time;
//
//	h_result = (int*)malloc(sizeof(int));
//
//	h_sudoku = readSudokuArray(filename);
//
//	//printArray(h_sudoku, N, N);
//
//	for(int i = 0; i < 9; i++)
//		for (int j = 0; j < 9; j++)
//			help[i][j] = h_sudoku[i][j];
//		
//
//	cudaMalloc((void **)&d_sudoku, N * N * sizeof(int));
//	cudaMalloc((void **)&d_number_presence, 243 * sizeof(int));
//	cudaMalloc((void **)&d_number_presence_nice_adding, 256 * sizeof(int));
//	cudaMalloc((void **)&d_result, sizeof(int));
//
//	cudaMemcpy(d_sudoku, help, (N*N)*sizeof(int), cudaMemcpyHostToDevice);
//
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//
//	cudaEventRecord(start, 0);
//	cudaEventRecord(stop, 0);
//
//	dim3 dimBlock = dim3(9, 9, 1);
//	dim3 dimGrid = dim3(1);
//	sharedMemorySize = 243 * sizeof(int);
//
//	checkCorrectness <<<dimGrid, dimBlock, sharedMemorySize>>> (d_sudoku, d_number_presence);
//
//	cudaDeviceSynchronize();
//
//	cudaMemcpy(h_number_presence, d_number_presence, 243 * sizeof(int), cudaMemcpyDeviceToHost);
//
//	for (int i = 0; i < 243; i++)
//		h_number_presence_nice_adding[i] = h_number_presence[i];
//
//	for (int i = 243; i < 256; i++)
//	{
//		h_number_presence_nice_adding[i] = 0;
//	}
//
//	cudaMemcpy(d_number_presence_nice_adding, h_number_presence_nice_adding, 256 * sizeof(int), cudaMemcpyHostToDevice);
//
//	addArray << <1, 256 >> > (d_number_presence_nice_adding, 256, d_result);
//	
//	cudaDeviceSynchronize();
//
//	cudaMemcpy(h_result, d_result, sizeof(int), cudaMemcpyDeviceToHost);
//	//cudaMemcpy(h_number_presence, d_number_presence, 243 * sizeof(int), cudaMemcpyDeviceToHost);
//	cudaMemcpy(h_number_presence_nice_adding, d_number_presence_nice_adding, 256 * sizeof(int), cudaMemcpyDeviceToHost);
//
//	cudaDeviceSynchronize();
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&time, start, stop);
//
//	
//	printf("Elapsed time: %f\n", time);
//
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	printOneDimArray(h_number_presence, 27, 9);
//	//for (int i = 0; i < 243; i++)
//		//printf("[%d : %d]", i, h_number_presence[i]);
//
//	printf("OBLICZONY WYNIK: %d", *h_result);
//
//	getchar();
//}
//
///*int* readSudokuArray_OneDimension(char* filename)
//{
//	int* h_sudoku = new int[9];
//
//	printf("SUDOKU FILENAME: %s\n", filename);
//	std::ifstream sudoku_file(filename);
//
//	int a0, a1, a2, a3, a4, a5, a6, a7, a8;
//	int i = 0;
//	
//	while (sudoku_file >> a0 >> a1 >> a2 >> a3 >> a4 >> a5 >> a6 >> a7 >> a8)
//	{
//		h_sudoku[i*9 + 0] = a0;
//		h_sudoku[i*9 + 1] = a1;
//		h_sudoku[i*9 + 2] = a2;
//		h_sudoku[i*9 + 3] = a3;
//		h_sudoku[i*9 + 4] = a4;
//		h_sudoku[i*9 + 5] = a5;
//		h_sudoku[i*9 + 6] = a6;
//		h_sudoku[i*9 + 7] = a7;
//		h_sudoku[i*9 + 8] = a8;
//		i++;
//	}
//	
//	return h_sudoku;
//	}*/