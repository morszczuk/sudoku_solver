#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <string>
#include <sstream>
#include <time.h>

#define SUD_SIZE 9

//reading sudoku quiz from a file
int** readSudokuArray(char* filename)
{
	int** h_sudoku = new int*[SUD_SIZE];

	for (int i = 0; i < SUD_SIZE; i++) {
		h_sudoku[i] = new int[SUD_SIZE];
	}

	//printf("SUDOKU FILENAME: %s\n", filename);
	std::ifstream sudoku_file(filename);

	int a0, a1, a2, a3, a4, a5, a6, a7, a8;
	int i = 0;

	while (sudoku_file >> a0 >> a1 >> a2 >> a3 >> a4 >> a5 >> a6 >> a7 >> a8)
	{
		h_sudoku[i][0] = a0;
		h_sudoku[i][1] = a1;
		h_sudoku[i][2] = a2;
		h_sudoku[i][3] = a3;
		h_sudoku[i][4] = a4;
		h_sudoku[i][5] = a5;
		h_sudoku[i][6] = a6;
		h_sudoku[i][7] = a7;
		h_sudoku[i][8] = a8;
		i++;
	}

	return h_sudoku;
}

//printing Array in sudoku-style.
void printArray(int** array, int N, int M)
{
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
			printf("%d |", array[i][j]);
		
		printf("\n");

		for (int j = 0; j < N; j++)
			printf("- |");
		
		printf("\n");
	}
}

__global__ void checkQuizFill(int d_quiz[SUD_SIZE][SUD_SIZE], int d_fill)
{
	int idx = blockDim.y*blockIdx.y + threadIdx.y;
	int idy = blockDim.x*blockIdx.x + threadIdx.x;

	] = d_quiz[idx][idy] > 0 ? 1 : 0;
}

hipError_t solveSudoku(int** h_sudoku_quiz)
{
	int *d_sudoku_quiz, *d_quiz_fill;
	int *

	hipMalloc((void **)&d_sudoku_quiz, SUD_SIZE * SUD_SIZE * sizeof(int));
	hipMalloc((void **)&d_quiz_fill, SUD_SIZE * SUD_SIZE * sizeof(int));

	hipMemcpy(d_sudoku_quiz, h_sudoku_quiz, SUD_SIZE * SUD_SIZE * sizeof(int), hipMemcpyHostToDevice);

	//int h_sudoku_quiz[SUD_SIZE][SUD_SIZE];

	//for(int i = 0; i < SUD_SIZE; i++)
	//	for (int j = 0; j < SUD_SIZE; j++)
	//		h_sudoku_quiz[i][j] = _sudoku_quiz[i][j];
}

int main()
{
	char filename[] = "arr_1_solved.txt";
	int ** h_sudoku_quiz;
	
	//RETRIEVING SUDOKU QUIZ
	h_sudoku_quiz = readSudokuArray(filename);
	printArray(h_sudoku_quiz, SUD_SIZE, SUD_SIZE);

	//STARTING TIME MEASURMENT
	clock_t begin = clock();
	
	//SOLVING SUDOKU 
	hipError_t cudaStatus = solveSudoku(h_sudoku_quiz);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "solveSudoku failed!");
		return 1;
	}

	//ENDING TIME MEASURMENT
	clock_t end = clock();
	printf("[FUNCTION TIME] %f ms\n", (double)(end - begin) / CLOCKS_PER_SEC * 1000);


	getchar();

	// RESETING CUDA DEVICE
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}